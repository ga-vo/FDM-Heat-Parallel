#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <vector>
#include "gnuplot-iostream.h"
#include <omp.h>
#include <chrono>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

// initial variables
int nc, nr, L, tmax, nx;
int nt = 10000;
int size = 0;
bool lleno = false;
double temp;
double dx, dt, alpha, r, r2, tempIzq, tempDer;

double t = 0;

double *u, *u_parallel;


__global__ void MDF_elem_op(double *uold, double *u, int nc_d, int nx_d, double r_d, double r2_d)
{
    int ar = threadIdx.x + blockIdx.x * blockDim.x;
    if(ar == 0){
        return;
    }
    if (ar + 1 < nx_d)
    {
        u[ar] = r_d * uold[ar - 1] + r2_d * uold[ar] + r_d * uold[ar + 1];
    }
    else
    {
        printf("ar: %d nx: %d \n", ar, nx_d);
    }
}

void MDF_parallel_cuda()
{
    long N = (nx-1);
    for (int m = 0; m < nt; m++)
    {
        t = t + dt;
        double *uold;
        double *u_d, *u_h;
        hipMalloc((void **)&u_d, nx * sizeof(double));
        hipMalloc((void **)&uold, nx * sizeof(double));
        hipMemcpy(u_d, u, nx * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(uold, u, nx * sizeof(double), hipMemcpyHostToDevice);

        u_h = (double *)malloc(nx * sizeof(double));
        MDF_elem_op<<<(N) / 256, 256>>>(uold, u_d, nc, nx, r, r2);
        hipMemcpy(u_h, u_d, nx * sizeof(double), hipMemcpyDeviceToHost);

        memcpy(u, u_h, nx * sizeof(double));
        hipFree(u_d);
        hipFree(uold);
        free(u_h);
    }
}

// Finite differences method
void MDF()
{
    // cout << u[nx - 1] << endl;
    // #pragma omp parallel for
    for (int m = 0; m < nt; m++)
    {
        // Create a copy of u array using memcpy (from C)
        double uold[nx];
        memcpy(uold, u, nx * sizeof(double));
        t = t + dt;

        for (int j = 0; j < nr; j++)
        {
            int ar = (j * nc);
            for (int i = 1; i < nc - 1; i++)
            {

                if (ar + 1 < nx)
                {
                    u[ar + i] = r * uold[ar + i - 1] + r2 * uold[ar + i] + r * uold[ar + i + 1];
                }
                else
                {
                    cout << "i: " << i << "  j: " << j << endl;
                }
            }
        }
    }
}

void MDF_parallel()
{
    // cout << u_parallel[nx - 1] << endl;

    for (int m = 0; m < nt; m++)
    {
        // Create a copy of u array using memcpy (from C)
        double uold[nx];
        memcpy(uold, u_parallel, nx * sizeof(double));
        t = t + dt;
#pragma omp parallel for
        for (int j = 0; j < nr; j++)
        {
            int ar = (j * nc);
            for (int i = 1; i < nc - 1; i++)
            {

                if (ar + 1 < nx)
                {
                    u_parallel[ar + i] = r * uold[ar + i - 1] + r2 * uold[ar + i] + r * uold[ar + i + 1];
                }
                else
                {
                    cout << "i: " << i << "  j: " << j << endl;
                }
            }
        }
    }
}

double *split(string s, int size)
{
    double *arr = new double[size];
    int occ = -1;
    int occ2 = 0;
#pragma omp parallel for
    for (int i = 1; i < size - 1; i++)
    {
        if (i % nc == 0)
        {
            arr[i] = tempIzq;
        }
        else
        {
            if (i % nc - 1 == 0)
            {
                arr[i] = tempDer;
            }
            else
            {
                occ2 = s.find(",", occ + 1);

                if (occ2 != string::npos)
                {
                    arr[i] = stod(s.substr(occ + 1, occ2));
                }
                else
                {
                    arr[i] = stod(s.substr(occ + 1, s.length() - 1));
                }
                occ = occ2;
            }
        }
    }
    lleno = true;

    return arr;
}

double *llenado(double temp, int size)
{
    double *arr = new double[size];

    for (int j = 0; j < nr; j++)
    {
        int ar = (j * nc);
        arr[ar] = tempIzq;
        if (!lleno)
        {
            for (int i = 1; i < nc - 1; i++)
            {

                arr[ar + i] = temp;
            }
        }
        else
        {
            for (int i = 1; i < nc - 1; i++)
            {

                arr[ar + i] = u[ar + i];
            }
        }
        arr[ar + nc - 1] = tempDer;
    }
    lleno = true;
    return arr;
}

void graph(string filename, string title)
{
    Gnuplot gp;
    ofstream salida(filename);

    // std::vector<double> pts_X;
    // std::vector<double> pts_Y;
    // std::vector<double> pts_Z;

    for (int j = 0; j < nr; j++)
    {
        int ar = (j * nc);
        for (int i = 0; i < nc; i++)
        {
            // pts_X.push_back(i);
            // pts_Y.push_back(j);
            // pts_Z.push_back(u[ar + i]);
            salida << i << " " << j << " " << u[ar + i] << endl;
        }
    }

    salida.close();

    // gp << "set xrang[0:" << max(nr,nc) << "]" << "\n set yrang[0:" << max(nr,nc) << "]" << endl;
    // gp << "plot '-' with image title '" << title << "'" << endl;
    // gp.send1d(make_tuple(pts_X, pts_Y, pts_Z));
    // gp.clearTmpfiles();
}

int main(int argc, char *argv[])
{
    if (argc > 1)
    {
        ifstream inputFile(argv[1]);
        string line;
        int nline = 1;
        double *arr;

        if (!inputFile)
        {
            cout << "Please use $ ./MDA <file> or verify filename" << endl;
            cout << "File format:\n     nrows\n     ncolumns\n     dt\n     dx\n     u\n     alpha(or k)" << endl;
        }
        else
        {

            while (getline(inputFile, line))
            {
                switch (nline)
                {
                case 1:
                    try
                    {
                        nr = stoi(line);
                    }
                    catch (invalid_argument e)
                    {
                        cout << "Please provide a valid file format: error in number of rows, provide a int" << endl;
                        exit(1);
                    }

                    break;
                case 2:
                    try
                    {
                        nc = stoi(line);
                    }
                    catch (invalid_argument e)
                    {
                        cout << "Please provide a valid file format: error in number of columns, provide a int" << endl;
                        exit(1);
                    }

                    break;
                case 3:
                    try
                    {
                        dt = stod(line);
                    }
                    catch (invalid_argument e)
                    {
                        cout << "Please provide a valid file format: error in number of columns, provide a int" << endl;
                        exit(1);
                    }

                    break;
                case 4:
                    try
                    {
                        dx = stod(line);
                    }
                    catch (invalid_argument e)
                    {
                        cout << "Please provide a valid file format: error in number of columns, provide a int" << endl;
                        exit(1);
                    }

                    break;
                case 5:
                    if (argc > 2)
                    {
                        if (!strcmp(argv[2], "--array"))
                        {
                            size = std::count(line.begin(), line.end(), ',') + 3;
                            if (size > 0)
                            {
                                arr = split(line, size);
                                u = arr;
                                nx = size;
                            }
                            else
                            {
                                cout << "Please provide a valid file format: error in U vector" << endl;
                            }
                        }
                    }
                    else
                    {
                        try
                        {
                            size = nr * nc;
                            nx = size;
                            temp = stod(line);
                        }
                        catch (invalid_argument e)
                        {
                            cout << "Please provide a valid file format: error in initial temperature, provide a float" << endl;
                            exit(1);
                        }
                    }
                    break;
                case 6:
                    try
                    {
                        tempIzq = stod(line);
                    }
                    catch (invalid_argument e)
                    {
                        cout << "Please provide a valid file format: error in number of columns, provide a int" << endl;
                        exit(1);
                    }

                    break;
                case 7:
                    try
                    {
                        tempDer = stod(line);
                    }
                    catch (invalid_argument e)
                    {
                        cout << "Please provide a valid file format: error in number of columns, provide a int" << endl;
                        exit(1);
                    }

                    break;
                case 8:
                    try
                    {
                        alpha = stod(line);
                    }
                    catch (invalid_argument e)
                    {
                        cout << "Please provide a valid file format: error in number of columns, provide a int" << endl;
                        exit(1);
                    }
                    break;
                }
                nline++;
            }
            u = llenado(temp, size);
            u[0] = tempIzq;
            u[nx - 1] = tempDer;
            u_parallel = new double[nx];
            memcpy(u_parallel, u, nx * sizeof(double));
            r = alpha * dt / (dx * dx);
            r2 = 1 - 2 * r;
            inputFile.close();
            cout << "nc: " << nc << "  nr: " << nr << "  nx: " << nx << "  dt: " << dt << "  dx: " << dx << "  alpha: " << alpha << endl;
            graph("input", "Estado Inicial");
            auto start = std::chrono::high_resolution_clock::now();
            MDF();
            auto stop = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
            // // graph("output", "Estado Final");
            std::cout << "Duracion Single-Core: " << duration.count() << std::endl;

            auto start1 = std::chrono::high_resolution_clock::now();
            MDF_parallel();
            auto stop1 = std::chrono::high_resolution_clock::now();
            auto duration1 = std::chrono::duration_cast<std::chrono::microseconds>(stop1 - start1);
            std::cout << "Duracion "<< omp_get_max_threads() << " Cores: " << duration1.count() << std::endl;

            auto startCuda = std::chrono::high_resolution_clock::now();
            MDF_parallel_cuda();
            auto stopCuda = std::chrono::high_resolution_clock::now();
            auto durationCuda = std::chrono::duration_cast<std::chrono::microseconds>(stopCuda - startCuda);
            std::cout << "Duracion CUDA: " << durationCuda.count() << std::endl;
            graph("output", "Estado Final");
            // std::cout << "Duracion " << omp_get_max_threads() << " cores: " << duration1.count() << std::endl;

            cout << "Finished :D" << endl;
        }
    }
    else
    {

        cout << "Please use $ ./MDA <file> " << endl;
        cout << "File format:\n     nrows\n     ncolumns\n     dt\n     dx\n     u\n     alpha(or k)" << endl;
    }

    return 0;
}